// %%cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Function to initialize a matrix with random values
void initializeMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = rand() % 10;
    }
}

// Function to print a matrix
void printMatrix(const int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

// CUDA kernel for element-wise matrix multiplication
__global__ void matrixMultiply(int *a, int *b, int *c, int n, int m, int p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < p) {
        int sum = 0;
        for (int i = 0; i < m; ++i) {
            sum += a[blockIdx.z * n * m + row * m + i] * b[blockIdx.z * m * p + i * p + col];
        }
        c[blockIdx.z * n * p + row * p + col] = sum;
    }
}

int main() {
    // Matrix dimensions
    int k = 13, N = 3, M = 3, P = 3;

    // Get matrix dimensions from the user
    /*
    printf("Enter the number of matrices (k): ");
    scanf("%d", &k);

    printf("Enter the number of rows for matrices A (N): ");
    scanf("%d", &N);

    printf("Enter the number of columns for matrices A and rows for matrices B (M): ");
    scanf("%d", &M);

    printf("Enter the number of columns for matrices B (P): ");
    scanf("%d", &P);
    */

    // Host matrices
    int *h_A, *h_B, *h_C;
    // Device matrices
    int *d_A, *d_B, *d_C;

    // Allocate memory on the host
    h_A = (int *)malloc(k * N * M * sizeof(int));
    h_B = (int *)malloc(k * M * P * sizeof(int));
    h_C = (int *)malloc(k * N * P * sizeof(int));

    // Initialize matrices with random values
    for (int i = 0; i < k; ++i) {
        initializeMatrix(&h_A[i * N * M], N, M);
        initializeMatrix(&h_B[i * M * P], M, P);
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, k * N * M * sizeof(int));
    hipMalloc((void**)&d_B, k * M * P * sizeof(int));
    hipMalloc((void**)&d_C, k * N * P * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, k * N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * M * P * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(10, 10);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (P + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       k); // Add k as the third dimension for handling multiple matrices

     // Record start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the CUDA kernel for element-wise matrix multiplication
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N, M, P);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time taken: %f milliseconds\n", milliseconds);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, k * N * P * sizeof(int), hipMemcpyDeviceToHost);

    // Print the matrices and result for each pair
    for (int i = 0; i < k; ++i) {
        printf("\nMatrix A%d:\n", i + 1);
        printMatrix(&h_A[i * N * M], N, M);

        printf("\nMatrix B%d:\n", i + 1);
        printMatrix(&h_B[i * M * P], M, P);

        printf("\nResult Matrix C%d (Multiplication of A%d and B%d):\n", i + 1, i + 1, i + 1);
        printMatrix(&h_C[i * N * P], N, P);
    }

    // Free allocated memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

